#include "hip/hip_runtime.h"
/*
 * MEX-CUDA GPU Parallel Computing Code for Long Short Term Memory Forward Process
 * This uses the MATLAB CUDA API in a MEX function that takes gpuArray inputs and 
 * returns gpuArray outputs.
 *
 * Written by Sungwook Wi, UMass
 */


#include "mex.h"
#include "gpu/mxGPUArray.h"

/*
 * Device code
 */
 
void __global__ lstm_forward_hidden(float * const Hall, float * const Call,
									float * const Gi, float * const Gf, float * const Go, float * const Gg,
									float const * const Ui, float const * const Uf, float const * const Uo, float const * const Ug,
									float const * const Wi, float const * const Wf, float const * const Wo, float const * const Wg,
									float const * const bi, float const * const bf, float const * const bo, float const * const bg,
									float const * const X, float * const H, float * const C,
									int const NumIn, int const NumHid, int const NumSeq, int const BatchSize)
{
	
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int co_x, co_y, j, k, N, M;

	N = NumHid * BatchSize;	
	M = NumIn * BatchSize;
	
	for (k = 0; k < NumSeq; k++) {		
		
		if (i < N) {	
			
			co_x = i % NumHid;
			co_y = i / NumHid;
			Gi[i] = 0.0;
			Gf[i] = 0.0;
			Go[i] = 0.0;
			Gg[i] = 0.0;
			for (j=0; j < NumIn; j++) {
				Gi[i] += Ui[NumHid*j+co_x] * X[NumIn*co_y+j+k*M];
				Gf[i] += Uf[NumHid*j+co_x] * X[NumIn*co_y+j+k*M];
				Go[i] += Uo[NumHid*j+co_x] * X[NumIn*co_y+j+k*M];
				Gg[i] += Ug[NumHid*j+co_x] * X[NumIn*co_y+j+k*M];
			}

			co_x = i % NumHid;
			co_y = i / NumHid;
			for (j=0; j < NumHid; j++) {
				Gi[i] += Wi[NumHid*j+co_x] * H[NumHid*co_y+j];
				Gf[i] += Wf[NumHid*j+co_x] * H[NumHid*co_y+j];
				Go[i] += Wo[NumHid*j+co_x] * H[NumHid*co_y+j];
				Gg[i] += Wg[NumHid*j+co_x] * H[NumHid*co_y+j];						
			}
			
			Gi[i] += bi[co_x];
			Gf[i] += bf[co_x];
			Go[i] += bo[co_x];
			Gg[i] += bg[co_x];
			
			Gi[i] = 1.0 / (1.0 + expf(-Gi[i]));
			Gf[i] = 1.0 / (1.0 + expf(-Gf[i]));
			Go[i] = 1.0 / (1.0 + expf(-Go[i]));
			Gg[i] = tanhf(Gg[i]);
			

			C[i] = Gg[i]*Gi[i] + Gf[i]*C[i]; 			
			H[i] = tanhf(C[i])*Go[i];
			
			Call[i+N*k] = C[i];
			Hall[i+N*k] = H[i];
			
		}  
		
	}		

	
}	

void __global__ lstm_forward_out(float * const Yhat, float const * const V, float const * const bv, float * const H, 
									int const NumOut, int const NumHid, int const BatchSize,
									float const * const drop_rate, float const * const drop_ind)
{
	
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int co_x, co_y, j, N;
	float drop_scale_factor;
	
	drop_scale_factor = 1-drop_rate[0];

	N = NumOut * BatchSize;
	
	if (i < N) {	 

		co_x = i % NumOut;
		co_y = i / NumOut;
		Yhat[i] = 0.0;
		for (j=0; j < NumHid; j++) {
			if (drop_ind[j] != 1) { 
				Yhat[i] += V[NumOut*j+co_x] * H[NumHid*co_y+j]/drop_scale_factor;	
			}	
			__syncthreads();	
		}
		Yhat[i] += bv[co_x];
		
	}																				
							
}

void __global__ lstm_forward_loss(float * const Yhat, float const * const Yobs, float const * const err_scale, float * const Loss, 
									int const NumOut, int const BatchSize)
{
	
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int N;

	N = NumOut * BatchSize;
	
	if (i < N) {	 
		Loss[i] = 0.5*pow(Yhat[i]-Yobs[i],2)*err_scale[i];
	}																				
	
}
 

/*
 * Host code
 */
void mexFunction(int nlhs, mxArray *plhs[],
                 int nrhs, mxArray const *prhs[])
{
    /* Declare all variables.*/
    // INPUTS
	mxGPUArray const *Ui;
	mxGPUArray const *Uf;
	mxGPUArray const *Uo;
	mxGPUArray const *Ug;
	mxGPUArray const *Wi;
	mxGPUArray const *Wf;
	mxGPUArray const *Wo;
	mxGPUArray const *Wg;
	mxGPUArray const *bi;
	mxGPUArray const *bf;
	mxGPUArray const *bo;
	mxGPUArray const *bg;
	mxGPUArray const *V;
	mxGPUArray const *bv;
	mxGPUArray const *X;
	mxGPUArray const *H;
	mxGPUArray const *C;		
	mxGPUArray const *Yobs;
	mxGPUArray const *err_scale;
	mxGPUArray const *dropRate;
	mxGPUArray const *dropInd;
    float const *d_Ui;
	float const *d_Uf;
	float const *d_Uo;
	float const *d_Ug;
	float const *d_Wi;
	float const *d_Wf;
	float const *d_Wo;
	float const *d_Wg;
	float const *d_bi;
	float const *d_bf;
	float const *d_bo;
	float const *d_bg;
	float const *d_V;
	float const *d_bv;
	float const *d_X;
	float *d_H;
	float *d_C;	
	float const *d_Yobs;
	float const *d_err_scale;
	float const *d_dropRate;
	float const *d_dropInd;
	
	// OUTPUTS
	mxGPUArray *Gi;
	mxGPUArray *Gf;
	mxGPUArray *Go;
	mxGPUArray *Gg;
	mxGPUArray *Call;
	mxGPUArray *Hall;
	mxGPUArray *Yhat;
	mxGPUArray *Loss;
	float *d_Gi;
	float *d_Gf;
	float *d_Go;
	float *d_Gg;
	float *d_Call;
	float *d_Hall;
	float *d_Yhat;
	float *d_Loss;

	
    int N;
    char const * const errId = "parallel:gpu:mexGPUExample:InvalidInput";
    char const * const errMsg1 = "Invalid input to MEX file: 20 inputs must be provided.";
    char const * const errMsg2 = "Invalid input to MEX file: Input(s) is not a GPU array.";
    char const * const errMsg3 = "Invalid input to MEX file: Input(s) is not a float array.";

    /* Choose a reasonably sized number of threads for the block. */
    int const threadsPerBlock = 1024;
    int blocksPerGrid;

    /* Initialize the MathWorks GPU API. */
    mxInitGPU();

    /* Throw an error if the input is not a GPU array. */
	if ((nrhs!=21)) {
        mexErrMsgIdAndTxt(errId, errMsg1);
    }

    if (!(mxIsGPUArray(prhs[0])) || 
		!(mxIsGPUArray(prhs[1])) ||
		!(mxIsGPUArray(prhs[2])) ||
		!(mxIsGPUArray(prhs[3])) ||
		!(mxIsGPUArray(prhs[4])) ||
		!(mxIsGPUArray(prhs[5])) ||
		!(mxIsGPUArray(prhs[6])) ||
		!(mxIsGPUArray(prhs[7])) ||
		!(mxIsGPUArray(prhs[8])) ||
		!(mxIsGPUArray(prhs[9])) ||
		!(mxIsGPUArray(prhs[10])) ||
		!(mxIsGPUArray(prhs[11])) ||
		!(mxIsGPUArray(prhs[12])) ||
		!(mxIsGPUArray(prhs[13])) ||
		!(mxIsGPUArray(prhs[14])) ||
		!(mxIsGPUArray(prhs[15])) ||
		!(mxIsGPUArray(prhs[16])) ||
		!(mxIsGPUArray(prhs[17])) ||
		!(mxIsGPUArray(prhs[18])) ||
		!(mxIsGPUArray(prhs[19])) ||
		!(mxIsGPUArray(prhs[20]))) 
    {
        mexErrMsgIdAndTxt(errId, errMsg2);
    }

	
    Ui = mxGPUCreateFromMxArray(prhs[0]);
	Uf = mxGPUCreateFromMxArray(prhs[1]);
	Uo = mxGPUCreateFromMxArray(prhs[2]);
	Ug = mxGPUCreateFromMxArray(prhs[3]);
	Wi = mxGPUCreateFromMxArray(prhs[4]);
	Wf = mxGPUCreateFromMxArray(prhs[5]);
	Wo = mxGPUCreateFromMxArray(prhs[6]);
	Wg = mxGPUCreateFromMxArray(prhs[7]);
	bi = mxGPUCreateFromMxArray(prhs[8]);
	bf = mxGPUCreateFromMxArray(prhs[9]);
	bo = mxGPUCreateFromMxArray(prhs[10]);
	bg = mxGPUCreateFromMxArray(prhs[11]);
	V = mxGPUCreateFromMxArray(prhs[12]);
	bv = mxGPUCreateFromMxArray(prhs[13]);
	X = mxGPUCreateFromMxArray(prhs[14]);	
	H = mxGPUCreateFromMxArray(prhs[15]);
	C = mxGPUCreateFromMxArray(prhs[16]);	
	Yobs = mxGPUCreateFromMxArray(prhs[17]);
	err_scale = mxGPUCreateFromMxArray(prhs[18]);
	dropRate = mxGPUCreateFromMxArray(prhs[19]);
	dropInd = mxGPUCreateFromMxArray(prhs[20]);
	

    /*
     * Verify that input really is a float array before extracting the pointer.
     */
    if ((mxGPUGetClassID(Ui) != mxSINGLE_CLASS) || 
    	(mxGPUGetClassID(Uf) != mxSINGLE_CLASS) ||
    	(mxGPUGetClassID(Uo) != mxSINGLE_CLASS) ||
    	(mxGPUGetClassID(Ug) != mxSINGLE_CLASS) ||
    	(mxGPUGetClassID(Wi) != mxSINGLE_CLASS) ||
    	(mxGPUGetClassID(Wf) != mxSINGLE_CLASS) ||
    	(mxGPUGetClassID(Wo) != mxSINGLE_CLASS) ||
    	(mxGPUGetClassID(Wg) != mxSINGLE_CLASS) ||
    	(mxGPUGetClassID(bi) != mxSINGLE_CLASS) ||
    	(mxGPUGetClassID(bf) != mxSINGLE_CLASS) ||
    	(mxGPUGetClassID(bo) != mxSINGLE_CLASS) ||
    	(mxGPUGetClassID(bg) != mxSINGLE_CLASS) ||
    	(mxGPUGetClassID(V)  != mxSINGLE_CLASS) ||
    	(mxGPUGetClassID(bv) != mxSINGLE_CLASS) ||
    	(mxGPUGetClassID(X)  != mxSINGLE_CLASS) ||
    	(mxGPUGetClassID(H)  != mxSINGLE_CLASS) ||
    	(mxGPUGetClassID(C)  != mxSINGLE_CLASS) ||
    	(mxGPUGetClassID(Yobs) != mxSINGLE_CLASS) ||
    	(mxGPUGetClassID(err_scale) != mxSINGLE_CLASS) ||
    	(mxGPUGetClassID(dropRate) != mxSINGLE_CLASS) ||
    	(mxGPUGetClassID(dropInd) != mxSINGLE_CLASS)) 
    {
        mexErrMsgIdAndTxt(errId, errMsg3);
    }
	
	

    /*
     * Now that we have verified the data type, extract a pointer to the input data on the device.
     */
	d_Ui = (float const *)(mxGPUGetDataReadOnly(Ui));
	d_Uf = (float const *)(mxGPUGetDataReadOnly(Uf));
	d_Uo = (float const *)(mxGPUGetDataReadOnly(Uo));
	d_Ug = (float const *)(mxGPUGetDataReadOnly(Ug));
	d_Wi = (float const *)(mxGPUGetDataReadOnly(Wi));
	d_Wf = (float const *)(mxGPUGetDataReadOnly(Wf));
	d_Wo = (float const *)(mxGPUGetDataReadOnly(Wo));
	d_Wg = (float const *)(mxGPUGetDataReadOnly(Wg));
	d_bi = (float const *)(mxGPUGetDataReadOnly(bi));
	d_bf = (float const *)(mxGPUGetDataReadOnly(bf));
	d_bo = (float const *)(mxGPUGetDataReadOnly(bo));
	d_bg = (float const *)(mxGPUGetDataReadOnly(bg));
	d_V = (float const *)(mxGPUGetDataReadOnly(V));
	d_bv = (float const *)(mxGPUGetDataReadOnly(bv));
	d_X = (float const *)(mxGPUGetDataReadOnly(X));	
	d_H = (float *)(mxGPUGetDataReadOnly(H));
	d_C = (float *)(mxGPUGetDataReadOnly(C));	
	d_Yobs = (float const *)(mxGPUGetDataReadOnly(Yobs));
	d_err_scale = (float const *)(mxGPUGetDataReadOnly(err_scale));
	d_dropRate = (float const *)(mxGPUGetDataReadOnly(dropRate));
	d_dropInd = (float const *)(mxGPUGetDataReadOnly(dropInd));


    /* Create a GPUArray to hold the result and get its underlying pointer. */
	const mwSize *dimsU = mxGPUGetDimensions(Ui);
	const mwSize *dimsX = mxGPUGetDimensions(X);	
	const mwSize *dimsV = mxGPUGetDimensions(V);
	size_t NumHid = dimsU[0];    // Number of hidden units
	size_t NumIn = dimsU[1];     // Number of input units (i.e., number of features)	
	size_t NumOut = dimsV[0];    // Number of output units	
	size_t NumSeq = dimsX[2];    // Sequence length
	size_t BatchSize = dimsX[1]; // Batch size (i.e., number of samples in a mini-batch)
	
	mwSize dims1[2] = {NumOut, BatchSize};
	mwSize dims2[2] = {NumHid, BatchSize};	
	mwSize dims3[3] = {NumHid, BatchSize, NumSeq};	

    Gi = mxGPUCreateGPUArray(2,dims2,mxGPUGetClassID(Ui),mxGPUGetComplexity(Ui),MX_GPU_DO_NOT_INITIALIZE);
	Gf = mxGPUCreateGPUArray(2,dims2,mxGPUGetClassID(Uf),mxGPUGetComplexity(Uf),MX_GPU_DO_NOT_INITIALIZE);
	Go = mxGPUCreateGPUArray(2,dims2,mxGPUGetClassID(Uo),mxGPUGetComplexity(Uo),MX_GPU_DO_NOT_INITIALIZE);
	Gg = mxGPUCreateGPUArray(2,dims2,mxGPUGetClassID(Ug),mxGPUGetComplexity(Ug),MX_GPU_DO_NOT_INITIALIZE);
	Call = mxGPUCreateGPUArray(3,dims3,mxGPUGetClassID(Ug),mxGPUGetComplexity(Ug),MX_GPU_DO_NOT_INITIALIZE);
	Hall = mxGPUCreateGPUArray(3,dims3,mxGPUGetClassID(Ug),mxGPUGetComplexity(Ug),MX_GPU_DO_NOT_INITIALIZE);
	Yhat = mxGPUCreateGPUArray(2,dims1,mxGPUGetClassID(Ug),mxGPUGetComplexity(Ug),MX_GPU_DO_NOT_INITIALIZE);
	Loss = mxGPUCreateGPUArray(2,dims1,mxGPUGetClassID(Ug),mxGPUGetComplexity(Ug),MX_GPU_DO_NOT_INITIALIZE);
    d_Gi = (float *)(mxGPUGetData(Gi));
	d_Gf = (float *)(mxGPUGetData(Gf));
	d_Go = (float *)(mxGPUGetData(Go));
	d_Gg = (float *)(mxGPUGetData(Gg));
	d_Call = (float *)(mxGPUGetData(Call));
	d_Hall = (float *)(mxGPUGetData(Hall));
	d_Yhat = (float *)(mxGPUGetData(Yhat));
	d_Loss = (float *)(mxGPUGetData(Loss));


    /*
     * Call the kernel using the CUDA runtime API. 
     * Here, I am using 1-d for grid and block deimension configuration.
     */

	N = (int)(NumHid * BatchSize);
    blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
	lstm_forward_hidden<<<blocksPerGrid, threadsPerBlock>>>(d_Hall, d_Call, 
															d_Gi, d_Gf, d_Go, d_Gg, 
															d_Ui, d_Uf, d_Uo, d_Ug, 
															d_Wi, d_Wf, d_Wo, d_Wg,
															d_bi, d_bf, d_bo, d_bg,
															d_X, d_H, d_C, 
															NumIn, NumHid, NumSeq, BatchSize);
					
	N = (int)(NumOut * BatchSize);
    blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;					
	lstm_forward_out<<<blocksPerGrid, threadsPerBlock>>>(d_Yhat, d_V, d_bv, d_H, NumOut, NumHid, BatchSize, d_dropRate, d_dropInd);	
	
	N = (int)(NumOut * BatchSize);
    blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
	lstm_forward_loss<<<blocksPerGrid, threadsPerBlock>>>(d_Yhat, d_Yobs, d_err_scale, d_Loss, NumOut, BatchSize);
	
	

    /* Wrap the result up as a MATLAB gpuArray for return. */  
	plhs[0] = mxGPUCreateMxArrayOnGPU(Yhat);
	plhs[1] = mxGPUCreateMxArrayOnGPU(Loss);
	plhs[2] = mxGPUCreateMxArrayOnGPU(Hall);
	plhs[3] = mxGPUCreateMxArrayOnGPU(Call);	
	
	

    /*
     * The mxGPUArray pointers are host-side structures that refer to device
     * data. These must be destroyed before leaving the MEX function.
     */
    // Freeing INPUT Array
	mxGPUDestroyGPUArray(Ui);
	mxGPUDestroyGPUArray(Uf);
	mxGPUDestroyGPUArray(Uo);
	mxGPUDestroyGPUArray(Ug);
	mxGPUDestroyGPUArray(Wi);
	mxGPUDestroyGPUArray(Wf);
	mxGPUDestroyGPUArray(Wo);
	mxGPUDestroyGPUArray(Wg);
	mxGPUDestroyGPUArray(bi);
	mxGPUDestroyGPUArray(bf);
	mxGPUDestroyGPUArray(bo);
	mxGPUDestroyGPUArray(bg);
    mxGPUDestroyGPUArray(X);	
    mxGPUDestroyGPUArray(H);	
	mxGPUDestroyGPUArray(C);
	mxGPUDestroyGPUArray(V);	
	mxGPUDestroyGPUArray(bv);
	mxGPUDestroyGPUArray(Yobs);
	mxGPUDestroyGPUArray(err_scale);
	mxGPUDestroyGPUArray(dropRate);
	mxGPUDestroyGPUArray(dropInd);
	
	// Freeing OUTPUY Array
	mxGPUDestroyGPUArray(Gi);
	mxGPUDestroyGPUArray(Gf);
	mxGPUDestroyGPUArray(Go);
	mxGPUDestroyGPUArray(Gg);
	mxGPUDestroyGPUArray(Call);
	mxGPUDestroyGPUArray(Hall);
	mxGPUDestroyGPUArray(Yhat);
	mxGPUDestroyGPUArray(Loss);

}
